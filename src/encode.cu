
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define SMAX 102
#define LMAX 15360
#include	<time.h>
#include	<sys/time.h>

//	Kernel	definition
// Decoder function which works on GPU side.
__global__ void decoder(char *cudalist, char *edited){

    int i = threadIdx.x;
    int j = blockIdx.x;
        if(*(cudalist+j*SMAX+i) == ','){
            *(edited+j*SMAX+i+1) = *(cudalist+j*SMAX+i+1);
            }
}
//Decoder function that works on CPU side.
void CpuDecoder(char *list, char *output){
    int i,j;
    for(i=0; i<LMAX; i++){
        for(j=0; j<SMAX; j++){
            if(*(list+i*SMAX+j) == ','){
               *(output+i*SMAX+j+1) = *(list+i*SMAX+j+1);
                //printf("%c", *(list+i*SMAX+j+1));
            }
        }
    }
}
//Prints the list. I used this function to test if my lists are empty or not.
void print(char *list){
    int i,j;
    for(i=0; i<LMAX; i++){
        for(j=0; j<SMAX-1; j++){
            printf("%c", *(list+i*SMAX+j));
        }
        printf("\n");
    }
}
//Writes "output" to the "decoded.txt" 
void writeToFile(char *list){

    FILE *fptr;
    if ((fptr = fopen("encoded.txt", "a+")) == NULL){
        printf("Error! opening file");
        exit(1);
    }
    int i;
    for (i=0; i<SMAX*LMAX; i++){
        if(*(list+i) != 0){
            fprintf(fptr,"%c",*(list+i));
        }
     }
    fclose(fptr);
}

//Read input file.
void readFromFile(char *list){
    int i = 0;
    char c[SMAX];
    FILE *fptr;
    if ((fptr = fopen("encodedfile.txt", "r")) == NULL){
        printf("Error! opening file");
        exit(1);
    }
    while(fgets(c, SMAX, fptr) != NULL){
        strtok(c, "\n");
        strcpy((list + i*SMAX), c);
        i++;
    }
    fclose(fptr);
}



int main()
{
    hipDeviceReset();
    char *list;
    char *cudalist;
    char *output;
    char *edited;

    list = (char*)malloc(LMAX * SMAX * sizeof(char));
    output = (char*)malloc(LMAX * SMAX * sizeof(char));

    hipMalloc((void **)&cudalist, LMAX * SMAX * sizeof(char));
    readFromFile(list);
    hipMalloc((void **)&edited, LMAX * SMAX * sizeof(char));


    struct	timeval	stop,	start;
    gettimeofday(&start,	NULL);
    hipMemcpy(cudalist, list, (LMAX * SMAX * sizeof(char)), hipMemcpyHostToDevice);
    decoder<<<LMAX,SMAX-2>>>(cudalist, edited);
    hipMemcpy(output, edited, (LMAX * SMAX * sizeof(char)), hipMemcpyDeviceToHost);
    gettimeofday(&stop,	NULL);
    float	elapsed	=	(stop.tv_sec	- start.tv_sec)	*	1000.0f	+	(stop.tv_usec	- start.tv_usec)	/	1000.0f;
    printf("Code	executed on GPU	in	%f	milliseconds.\n",	elapsed);


    gettimeofday(&start,	NULL);
    CpuDecoder(list, output);
    gettimeofday(&stop,	NULL);
    elapsed	=	(stop.tv_sec	- start.tv_sec)	*	1000.0f	+	(stop.tv_usec	- start.tv_usec)	/	1000.0f;
    printf("Code	executed on CPU	in	%f	milliseconds.\n",	elapsed);



    writeToFile(output);
    free(list);
    free(output);
    hipFree(cudalist);
    hipFree(edited);
    return -1;
}



